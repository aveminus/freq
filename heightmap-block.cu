#include "hip/hip_runtime.h"
#include <stdio.h>
#include "heightmap-block.cu.h"

__global__ void kernel_merge(
                cudaPitchedPtrType<float> inBlock,
                cudaPitchedPtrType<float> outBlock,
                float resample_width,
                float resample_height,
                float in_offset,
                float out_offset,
                float in_valid_samples)
{
    elemSize3_t writePos;
    if( !outBlock.unwrapCudaGrid( writePos ))
        return;

    float val = 0;
    unsigned n = 0;

    if (writePos.x>=out_offset)
    {
        for (float x = 0; x < resample_width; x++)
        {
            float s = in_offset + x + resample_width*(writePos.x-out_offset);
            if ( s >= in_offset + in_valid_samples + .25f*resample_width )
                x=resample_width;
            else for (float y = 0; y < resample_height; y++)
            {
                float t = y + resample_height*writePos.y;

                elemSize3_t readPos = make_elemSize3_t( s, t, 0 );
                if ( inBlock.valid(readPos) ) {
                    val += inBlock.elem(readPos);

                    //outBlock.e( writePos ) = val;
                    //return;

                    n ++;
                }
            }
        }
    }

    if (0<n) {
        val/=n;
        outBlock.elem( writePos ) = val;
    }
}

extern "C"
void blockMerge( cudaPitchedPtrType<float> inBlock,
                 cudaPitchedPtrType<float> outBlock,
                 float in_sample_rate,
                 float out_sample_rate,
                 float in_frequency_resolution,
                 float out_frequency_resolution,
                 float in_offset,
                 float out_offset,
                 float in_valid_samples,
                 unsigned hip_stream)
{
    dim3 grid, block;
    unsigned block_size = 128;

    outBlock.wrapCudaGrid2D( block_size, grid, block );

    float resample_width = in_sample_rate/out_sample_rate;
    float resample_height = in_frequency_resolution/out_frequency_resolution;

    kernel_merge<<<grid, block, hip_stream>>>(
        inBlock, outBlock,
        resample_width,
        resample_height,
        in_offset, out_offset, in_valid_samples );
}

texture<float2, 1, hipReadModeElementType> chunkTexture;

/**
 kernel_merge_chunk has one thread for each output element
*/
__global__ void kernel_merge_chunk(
                cudaPitchedPtrType<float2> inChunk,
                cudaPitchedPtrType<float> outBlock,
                float resample_width,
                float resample_height,
                float in_offset,
                float out_offset,
                unsigned in_count )
{
    elemSize3_t writePos;
    if( !outBlock.unwrapCudaGrid( writePos ))
        return;

    float val = 0;
    float n = 0;

    if (writePos.x>=out_offset)
    {
        // TODO xs should depend on hz
        float ff = writePos.y/((float)outBlock.getNumberOfElements().y - 1);

        float xs = 2.f/(ff*ff);//resample_width/10;
        if (1>xs) xs=1;
        for (float x = 0; x < resample_width; x+=xs)
        {
            float s = in_offset + x + resample_width*(writePos.x-out_offset);

            if ( s > in_offset + in_count + .25f*resample_width)
                x=resample_width; // abort for x loop, faster than "break;"
            else for (float y = 0; y < resample_height; y++)
            {
                //float y = 0;
                float t = y + resample_height*writePos.y;

                elemSize3_t readPos = make_elemSize3_t( s, t, 0 );
                readPos = inChunk.clamp(readPos);
                if ( inChunk.valid(readPos) ) {
                    float ff = t/(float)inChunk.getNumberOfElements().y;
                    float if0 = 40.f/(2.0f + 35*ff*ff*ff);

                    //float2 c = inChunk.elem(readPos);
                    float2 c = tex1Dfetch(chunkTexture, inChunk.eOffs(readPos));
                    val = max(val, if0*sqrt(if0*(c.x*c.x + c.y*c.y)));

 //outBlock.e( writePos ) = 4*val;
 //return;
/*
  TODO use command line argument "yscale"
                        case Yscale_Linear:
                            v[2][df] = amplitude;
                            break;
                        case Yscale_ExpLinear:
                            v[2][df] = amplitude * exp(.001*fi);
                            break;
                        case Yscale_LogLinear:
                            v[2][df] = amplitude;
                            v[2][df] = log(1+fabsf(v[2][df]))*(v[2][df]>0?1:-1);
                            break;
                        case Yscale_LogExpLinear:
                            v[2][df] = amplitude * exp(.001*fi);
                            v[2][df] = log(1+fabsf(v[2][df]))*(v[2][df]>0?1:-1);
                            */

                    n++;
                }
            }
        }
    }
/*
    __syncthreads();
*/
    if (0<n) {
        //val/=n;
        outBlock.e( writePos ) = val;
    }
}

#define BLOCK_SIZE 64 // manally adjusted to increase performance

struct read_params {
    unsigned
        block_first_read,
        block_last_read,
        num_reads,
        thread_first_read,
        thread_last_read,
        start_y,
        end_y;
};

__device__ __host__ read_params computeReadParams( unsigned in_offset, unsigned in_count, float out_offset,
                                                   float resample_height, float resample_width,
                                                   elemSize3_t writePos, bool firstThreadInBlock )
{
    read_params p;

    float threadFirstWrite = writePos.x;
    float threadLastWrite = writePos.x+1;

    // Here, BLOCK_SIZE is the number of elements that each block is responsible for writing
    float blockFirstWrite = writePos.x/BLOCK_SIZE*BLOCK_SIZE; // integer division
    float blockLastWrite = blockFirstWrite + BLOCK_SIZE;

    // Don't write anything before out_offset, if the entire range is before out_offset the range will be
    // [out_offset, out_offset[ which is an empty set. However, [blockFirstWrite, blockLastWrite[ might be
    // non-empty even though [myFirstWrite, myLastWrite[ is empty.
    if (threadFirstWrite < out_offset) threadFirstWrite = out_offset;
    if (threadLastWrite  < out_offset) threadLastWrite  = out_offset;
    if (blockFirstWrite  < out_offset) blockFirstWrite  = out_offset;
    if (blockLastWrite   < out_offset) blockLastWrite   = out_offset;

    p.thread_first_read = in_offset + (threadFirstWrite - out_offset) * resample_width;
    p.thread_last_read  = in_offset + (threadLastWrite  - out_offset) * resample_width;

    if (p.thread_first_read < in_offset)          p.thread_first_read = in_offset;
    if (p.thread_last_read  > in_offset+in_count) p.thread_last_read  = in_offset+in_count;

    if (firstThreadInBlock)
    {
        p.block_first_read = in_offset + (blockFirstWrite - out_offset) * resample_width;
        p.block_last_read  = in_offset + (blockLastWrite  - out_offset) * resample_width;

        if (p.block_first_read < in_offset)          p.block_first_read = in_offset;
        if (p.block_last_read  > in_offset+in_count) p.block_last_read  = in_offset+in_count;

        // Here, BLOCK_SIZE is the number of elements that a block reads in each read chunk
        p.block_first_read =  p.block_first_read              /BLOCK_SIZE; // integer division
        p.block_last_read  = (p.block_last_read+ BLOCK_SIZE-1)/BLOCK_SIZE;

        if (p.block_first_read<p.block_last_read)
            p.num_reads = p.block_last_read-p.block_first_read;
        else
            p.num_reads = 0;

        p.block_first_read *= BLOCK_SIZE;
        p.block_last_read  *= BLOCK_SIZE;

        p.start_y = resample_height *  writePos.y;
        p.end_y   = resample_height * (writePos.y+1);
        if (p.end_y == p.start_y) p.end_y++;
    }

    return p;
}

/**
    kernel_merge_chunk2 has one thread for each output element.
    Threads collaborate in warps to serve eachother. Each block only have one warp.

    @param resample_width is the number of input elements that should be taken into
           account for the output element that each thread is to write.
*/
__global__ void kernel_merge_chunk2(
                cudaPitchedPtrType<float2> inChunk,
                cudaPitchedPtrType<float> outBlock,
                float resample_width,
                float resample_height,
                unsigned in_offset,
                float out_offset,
                unsigned in_count,
                Heightmap::TransformMethod transformMethod )
{
/**
    Merging like this for resample_width = 3, resample_height=1, in_offset=1, out_offset=0
    chunk: 1|1234575652165761|9682657451433|321
    out:    |  5  7  5  7  9 | 8  7  5  4  | => |57576|8754|

    That is; discard element that begins before outBlock but don't discard fraction of element that end after outBlock.

    If an element would include more than BLOCK_SIZE, the last elements are discarded.
*/
    __shared__ float val[BLOCK_SIZE];

    elemSize3_t writePos = make_elemSize3_t(
                            __umul24(blockIdx.x, blockDim.x) + threadIdx.x,
                            __umul24(blockIdx.y, blockDim.y) + threadIdx.y,
                            __umul24(blockIdx.z, blockDim.z) + threadIdx.z);
    //outBlock.elem(writePos) = 0;
    //return;

    read_params thread_p = computeReadParams( in_offset, in_count, out_offset,
                                       resample_height, resample_width,
                                       writePos, 0==threadIdx.x );

    __shared__ read_params block_read;
    if (0==threadIdx.x)
        block_read = thread_p;

    float myVal = -1; // Negative value that indicates that no value was fetched

    __syncthreads();

    if (0 != block_read.num_reads) for (unsigned y = block_read.start_y; y < block_read.end_y; y++)
    {
        float ff = y/(float)inChunk.getNumberOfElements().y;
        float if0 = 20.f/(2.0f + 35.f*ff*ff*ff);
        if0=if0*if0*if0;

        for (unsigned i=0; i<block_read.num_reads; i++)
        {
            unsigned base = block_read.block_first_read + i*BLOCK_SIZE;
            elemSize3_t readPos = make_elemSize3_t( base + threadIdx.x, y, 0 );

            bool valid = readPos.x >= in_offset &&
                         readPos.x < in_offset + in_count &&
                         inChunk.valid(readPos);

            // Read from global memory
            float2 c = valid ? inChunk.elem(readPos) : make_float2(0,0);

            if (transformMethod==Heightmap::TransformMethod_Cwt_phase)
                val[threadIdx.x] = 0.1f*(M_PI + atan2(c.y, c.x))*(1.f/(2*M_PI));
            else
                val[threadIdx.x] = if0*(c.x*c.x + c.y*c.y);

            __syncthreads();

            if (0) { // Each thread minds its own business
                if ( thread_p.thread_last_read  > base &&
                     thread_p.thread_first_read < thread_p.thread_last_read &&
                     thread_p.thread_first_read < base+BLOCK_SIZE )
                {
                    unsigned start = max(thread_p.thread_first_read, base) - base;
                    unsigned end = min(thread_p.thread_last_read, base + BLOCK_SIZE) - base;
    //                myVal = val[start];
                    for (unsigned x = start; x<end; x++)
                        myVal = max(myVal, val[x]);
                }
            } else {
                // Find the maxima in all of val and use that, half of the threads go to sleep at each step
                // This is not as exact as "Each thread minds its own business" but differences are rarely visible.
                if (128==BLOCK_SIZE) {
                    if (threadIdx.x<64) val[threadIdx.x] = max(val[threadIdx.x], val[64 + threadIdx.x]);
                    if (threadIdx.x<32) val[threadIdx.x] = max(val[threadIdx.x], val[32 + threadIdx.x]);
                }
                if (128==BLOCK_SIZE || 32==BLOCK_SIZE) {
                    if (threadIdx.x<16) val[threadIdx.x] = max(val[threadIdx.x], val[16 + threadIdx.x]);
                    if (threadIdx.x<8) val[threadIdx.x] = max(val[threadIdx.x], val[8 + threadIdx.x]);
                    if (threadIdx.x<4) val[threadIdx.x] = max(val[threadIdx.x], val[4 + threadIdx.x]);
                    if (threadIdx.x<2) val[threadIdx.x] = max(val[threadIdx.x], val[2 + threadIdx.x]);
                    if (threadIdx.x<1) val[0] = max(val[0], val[1]);
                }
                if (64==BLOCK_SIZE) {
                    if (threadIdx.x<16) {
                        val[threadIdx.x] = max(max(val[threadIdx.x], val[48 + threadIdx.x]),
                                               max(val[32 + threadIdx.x], val[16 + threadIdx.x]));

                    if (threadIdx.x<4) {
                        val[threadIdx.x] = max(max(val[threadIdx.x], val[4 + threadIdx.x]),
                                               max(val[8 + threadIdx.x], val[12 + threadIdx.x]));

                    if (threadIdx.x<1) {
                        val[0] = max(max(val[0], val[1]), max(val[2], val[3]));
                    }}}
                }

                __syncthreads();

                if ( thread_p.thread_last_read  > base &&
                     thread_p.thread_first_read < thread_p.thread_last_read &&
                     thread_p.thread_first_read < base+BLOCK_SIZE )
                {
                    myVal = max(myVal, val[0]);
                }
            }
        }
    }

    if (outBlock.valid( writePos ) && 0<=myVal)
    {
        if (transformMethod!=Heightmap::TransformMethod_Cwt_phase)
            myVal = sqrtf(myVal);

        outBlock.elem( writePos ) = myVal;
    }
}

extern "C"
void blockMergeChunk( cudaPitchedPtrType<float2> inChunk,
                 cudaPitchedPtrType<float> outBlock,
                 float in_sample_rate,
                 float out_sample_rate,
                 float in_frequency_resolution,
                 float out_frequency_resolution,
                 unsigned in_offset,
                 float out_offset,
                 unsigned in_count,
                 Heightmap::TransformMethod transformMethod,
                 unsigned hip_stream)
{
    unsigned block_size;

    const unsigned version = 2;
    switch (version)
    {   case 1: block_size = 128; break;
        case 2: block_size = BLOCK_SIZE; break;
    }

    // For version 2
    dim3 block(block_size,1,1);
    uint3 nElems = outBlock.getNumberOfElements();

    // Limit kernel size
    {
        unsigned last_write = ceil(out_offset + in_count*out_sample_rate/in_sample_rate);
        nElems.x = min(nElems.x, last_write);
        unsigned unused = out_offset/block_size;
        unused*=block_size;
        // printf("unused = %u\n", unused);
        nElems.x -= unused;
        out_offset -= unused;
        hipPitchedPtr cpp = outBlock.getCudaPitchedPtr();
        cpp.ptr = &((float*)cpp.ptr)[unused];
        outBlock =  cudaPitchedPtrType<float>(cpp);
    }

    dim3 grid;

    switch (version)
    {   case 1: outBlock.wrapCudaGrid2D(block_size, grid, block); break;
        case 2: grid = make_uint3(
                    int_div_ceil(nElems.x, block.x),
                    int_div_ceil(nElems.y, block.y),
                    int_div_ceil(nElems.z, block.z));
            break;
    }

    float resample_width = in_sample_rate/out_sample_rate;
    float resample_height = (in_frequency_resolution+2)/out_frequency_resolution;

    if (0) {
        elemSize3_t sz_o = outBlock.getNumberOfElements();
        elemSize3_t sz_i = inChunk.getNumberOfElements();
        //fprintf(stdout,"sz_o (%d, %d, %d)\tsz_i (%d, %d, %d)\n", sz_o.x, sz_o.y, sz_o.z, sz_i.x, sz_i.y, sz_i.z );


        fprintf(stdout,"\ngrid (%d, %d, %d)\tblock (%d, %d, %d)\n", grid.x, grid.y, grid.z, block.x, block.y, block.z );
        fprintf(stdout,"(in sr %g, out sr %g, w=%g) (in f %g, out f %g, h=%g)\n\tin o %u, out o %g, in count %u\n",
            in_sample_rate, out_sample_rate, resample_width,
            in_frequency_resolution, out_frequency_resolution, resample_height,
            in_offset, out_offset, in_count);
        fprintf(stdout,"outBlock(%d,%d,%d) pitch %lu\n",
            outBlock.getNumberOfElements().x,
            outBlock.getNumberOfElements().y,
            outBlock.getNumberOfElements().z,
            outBlock.getCudaPitchedPtr().pitch );
        fprintf(stdout,"inChunk(%d,%d,%d) pitch %lu\n",
            inChunk.getNumberOfElements().x,
            inChunk.getNumberOfElements().y,
            inChunk.getNumberOfElements().z,
            inChunk.getCudaPitchedPtr().pitch );
        fflush(stdout);

    }

    if (0)
    {
        unsigned x_count = grid.x*block.x;
        printf("nElems.x = %u, x_count = %u\n", nElems.x, x_count);

        if(1) for (int x=0; x<x_count; x++)
        {
            elemSize3_t writePos = make_elemSize3_t(x,0,0);
            elemSize3_t myThreadIdx = make_elemSize3_t(0,0,0);

            read_params p = computeReadParams( in_offset, in_count, out_offset,
                                               resample_height, resample_width,
                                               writePos, 0==myThreadIdx.x );

            printf("%3u [%u, %u](%u) > [%u, %u]\n", x,
                p.block_first_read, p.block_last_read, p.num_reads,
                p.thread_first_read, p.thread_last_read);
        }
    }

    switch (version)
    {
    case 1:
        {
            hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float2>();
            hipBindTexture(0, &chunkTexture, inChunk.ptr(), &channelDesc, inChunk.getCudaPitchedPtr().pitch * inChunk.getCudaPitchedPtr().ysize);

            kernel_merge_chunk<<<grid, block, hip_stream>>>(
                inChunk, outBlock,
                resample_width,
                resample_height,
                in_offset, out_offset, in_count );
            hipUnbindTexture(&chunkTexture);
            break;
        }
    case 2:
        {
            kernel_merge_chunk2<<<grid, block, hip_stream>>>(
                inChunk, outBlock,
                resample_width,
                resample_height,
                in_offset, out_offset, in_count, transformMethod );
            break;
        }
    }

    hipUnbindTexture(&chunkTexture);
}

__global__ void kernel_expand_stft(
                cudaPitchedPtrType<float2> inStft,
                cudaPitchedPtrType<float> outBlock,
                float start,
                float steplogsize,
                float out_offset,
                float out_length )
{
    // Element number
    const unsigned
            y = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;

    unsigned nFrequencies = outBlock.getNumberOfElements().y;
    if( y >= nFrequencies )
        return;

    float ff = y/(float)nFrequencies;
    float hz_out = start*exp(ff*steplogsize);

    float max_stft_hz = 44100.f/2;
    float min_stft_hz = 44100.f/(2*inStft.getNumberOfElements().x);
    float read_f = max(0.f,min(1.f,(hz_out-min_stft_hz)/(max_stft_hz-min_stft_hz)));

    float2 c;

    float p = read_f*inStft.getNumberOfElements().x;
    elemSize3_t readPos = make_elemSize3_t( p, 0, 0 );
    inStft.clamp(readPos);
    c = inStft.elem(readPos);
    float val1 = sqrt(c.x*c.x + c.y*c.y);

    readPos.x++;
    inStft.clamp(readPos);
    c = inStft.elem(readPos);
    float val2 = sqrt(c.x*c.x + c.y*c.y);

    p-=(unsigned)p;
    float val = .02f*(val1*(1-p)+val2*p);
    const float f0 = 2.0f + 35*ff*ff*ff;
    val*=f0;

    elemSize3_t writePos = make_elemSize3_t( 0, y, 0 );
    for (writePos.x=out_offset; writePos.x<out_offset + out_length && writePos.x<outBlock.getNumberOfElements().x;writePos.x++)
    {
        outBlock.e( writePos ) = val;
    }
}


extern "C"
void expandStft( cudaPitchedPtrType<float2> inStft,
                 cudaPitchedPtrType<float> outBlock,
                 float min_hz,
                 float max_hz,
                 float out_offset,
                 float out_length,
                 unsigned hip_stream)
{
    dim3 block(256,1,1);
    dim3 grid( int_div_ceil(outBlock.getNumberOfElements().y, block.x), 1, 1);

    if(grid.x>65535) {
        printf("====================\nInvalid argument, number of floats in complex signal must be less than 65535*256.\n===================\n");
        return;
    }

    float start = min_hz/2;
    float steplogsize = log(max_hz)-log(min_hz);

    kernel_expand_stft<<<grid, block, hip_stream>>>(
        inStft, outBlock,
        start,
        steplogsize,
        out_offset,
        out_length );
}


__global__ void kernel_expand_complete_stft(
                cudaPitchedPtrType<float> inStft,
                cudaPitchedPtrType<float> outBlock,
                float start,
                float steplogsize,
                float out_stft_size,
                float out_offset,
                float in_min_hz,
                float in_max_hz,
                unsigned in_stft_size)
{
    // Element number
    const unsigned
            x = __umul24(blockIdx.x,blockDim.x) + threadIdx.x,
            y = __umul24(blockIdx.y,blockDim.y) + threadIdx.y;

    float val;
    /*if (1 || 0==threadIdx.x)*/ {
            unsigned nFrequencies = outBlock.getNumberOfElements().y;
        if( y >= nFrequencies )
            return;

        float ff = y/(float)nFrequencies;
        float hz_out = start*exp(ff*steplogsize);

        float read_f = max(0.f,min(1.f,(hz_out-in_min_hz)/(in_max_hz-in_min_hz)));

        float2 c;
        float q = max(0.f, (x+out_offset)/out_stft_size);
        unsigned chunk = (unsigned)q;
        q-=chunk;
        float p = ((chunk+read_f)*in_stft_size);
        unsigned read_start = ((unsigned)p)*2;
        p-=(unsigned)p;

        c.x = inStft.elem(make_elemSize3_t( read_start, 0, 0 ));
        c.y = inStft.elem(make_elemSize3_t( read_start+1, 0, 0 ));
        float val1 = sqrt(c.x*c.x + c.y*c.y);

        c.x = inStft.elem(make_elemSize3_t( read_start+2*in_stft_size, 0, 0 ));
        c.y = inStft.elem(make_elemSize3_t( read_start+2*in_stft_size+1, 0, 0 ));
        float val2 = sqrt(c.x*c.x + c.y*c.y);

        unsigned read_secondline = min(read_start+2, 2*((1+chunk)*in_stft_size-1));
        c.x = inStft.elem(make_elemSize3_t( read_secondline, 0, 0 ));
        c.y = inStft.elem(make_elemSize3_t( read_secondline+1, 0, 0 ));
        float val3 = sqrt(c.x*c.x + c.y*c.y);

        c.x = inStft.elem(make_elemSize3_t( read_secondline+2*in_stft_size, 0, 0 ));
        c.y = inStft.elem(make_elemSize3_t( read_secondline+2*in_stft_size+1, 0, 0 ));
        float val4 = sqrt(c.x*c.x + c.y*c.y);

        // Perform a kind of bicubic interpolation
        p = 3*p*p-2*p*p*p;
        q = 3*q*q-2*q*q*q;
        val = .07f*((val1*(1-q)+val2*q)*(1-p) + (val3*(1-q)+val4*q)*p);

        const float f0 = 2.0f + 35*ff*ff*ff;
        val*=sqrt(f0);

        //float if0 = 40.f/(2.0f + 35*ff*ff*ff);
        //float if0 = 40.f/(2.0f + 35.f*ff*ff*ff);
        //if0=if0*if0*if0;
        //val=sqrt(if0*val);

        val*=19.f;
    }

    val /= in_stft_size;

    elemSize3_t writePos = make_elemSize3_t( x, y, 0 );
    outBlock.e( writePos ) = val;
}


extern "C"
void expandCompleteStft( cudaPitchedPtrType<float> inStft,
                 cudaPitchedPtrType<float> outBlock,
                 float out_min_hz,
                 float out_max_hz,
                 float out_stft_size,
                 float out_offset,
                 float in_min_hz,
                 float in_max_hz,
                 unsigned in_stft_size,
                 unsigned hip_stream)
{
    dim3 block(32,1,1);
    dim3 grid( outBlock.getNumberOfElements().x/block.x, outBlock.getNumberOfElements().y, 1);

    if(grid.x>65535 || grid.y>65535 || 0!=(in_stft_size%32)) {
        printf("====================\nInvalid argument, expandCompleteStft.\n===================\n");
        return;
    }

    float start = out_min_hz/2;
    float steplogsize = log(out_max_hz)-log(out_min_hz);

    kernel_expand_complete_stft<<<grid, block, hip_stream>>>(
        inStft, outBlock,
        start,
        steplogsize,
        out_stft_size,
        out_offset,
        in_min_hz,
        in_max_hz,
        in_stft_size );
}
