#include "hip/hip_runtime.h"
#include "splinefilter.cu.h"
#include <resamplecuda.cu.h>
#include <operate.cu.h>
#include "cudaglobalstorage.h"

template<typename Reader>
class Spliner
{
public:
    Spliner(Reader reader, unsigned N, bool save_inside)
        :   reader(reader),
            N(N),
            save_inside(save_inside)
    {}


    __device__ void operator()(float2& e, ResamplePos const& v)
    {
        // Count the number of times a line from v to infinity crosses the spline

        // Walk along +y axis only
        bool inside = false;
        float mindisty = FLT_MAX;
        float mindistx = FLT_MAX;
        for (unsigned i=0; i<N; ++i)
        {
            unsigned j = (i+1)%N;
            float2 p = reader(i), q = reader(j);
            float r = (v.x - p.x)/(q.x - p.x);
            if (0 <= r && 1 > r)
            {
                float y = p.y + (q.y-p.y)*r;
                if (y > v.y)
                {
                    inside = !inside;
                }
                if (mindisty > fabsf(y-v.y))
                    mindisty = fabsf(y-v.y);
            }
            r = (v.y - p.y)/(q.y - p.y);
            if (0 <= r && 1 > r)
            {
                float x = p.x + (q.x-p.x)*r;
                if (mindistx > fabsf(x-v.x))
                    mindistx = fabsf(x-v.x);
            }
        }

        if (inside != save_inside)
        {
            float d = 1 - min(mindisty*(1/1.f), mindistx*(1/4.f));
            if (d < 0)
                d = 0;

            float2 f = e;
            e = make_float2( f.x*d, f.y*d );
        }
    }


private:
    Reader reader;
    unsigned N;
    bool save_inside;
};


void applyspline(
        Tfr::ChunkData::Ptr datap,
        DataStorage<Tfr::ChunkElement>::Ptr splinep, bool save_inside )
{
    cudaPitchedPtrType<float2> data( CudaGlobalStorage::ReadWrite<2>(datap).getCudaPitchedPtr());
    cudaPitchedPtrType<float2> spline( CudaGlobalStorage::ReadOnly<1>(splinep).getCudaPitchedPtr());

    Spliner< Read1D<float2> > spliner(
            Read1D_Create<float2>( spline ),
            spline.getNumberOfElements().x,
            save_inside );

    elemSize3_t sz = data.getNumberOfElements();
    element_operate<float2>( data, make_float4(0, 0, sz.x, sz.y), spliner );

    Read1D_UnbindTexture<float2>();
}
