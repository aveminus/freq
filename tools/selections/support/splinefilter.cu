#include "hip/hip_runtime.h"
#include "splinefilter.cu.h"
#include <operate.cu.h>
#include <resample.cu.h>

template<typename Reader>
class Spliner
{
public:
    Spliner(Reader reader, unsigned N, bool save_inside)
        :   reader(reader),
            N(N),
            save_inside(save_inside)
    {}


    __device__ void operator()(float2& e, float2 const& v)
    {
        // Count the number of times a line from v to infinity crosses the spline

        // Walk along +y axis only
        bool inside = false;
        float mindisty = FLT_MAX;
        float mindistx = FLT_MAX;
        for (unsigned i=0; i<N; ++i)
        {
            unsigned j = (i+1)%N;
            float2 p = reader(make_uint2(i,0)), q = reader(make_uint2(j,0));
            float r = (v.x - p.x)/(q.x - p.x);
            if (0 <= r && 1 > r)
            {
                float y = p.y + (q.y-p.y)*r;
                if (y > v.y)
                {
                    inside = !inside;
                }
                if (mindisty > fabsf(y-v.y))
                    mindisty = fabsf(y-v.y);
            }
            r = (v.y - p.y)/(q.y - p.y);
            if (0 <= r && 1 > r)
            {
                float x = p.x + (q.x-p.x)*r;
                if (mindistx > fabsf(x-v.x))
                    mindistx = fabsf(x-v.x);
            }
        }

        // TODO soft edges

        if (inside != save_inside)
        {
            float d = 1 - min(mindisty*(1/1.f), mindistx*(1/4.f));
            if (d < 0)
                d = 0;
            //mindisty = 0.5;

            e.x *= d;
            e.y *= d;
        }
    }


private:
    Reader reader;
    unsigned N;
    bool save_inside;
};


void applyspline(
        cudaPitchedPtrType<float2> data,
        cudaPitchedPtrType<float2> spline, bool save_inside )
{
    bindtex<float2>( spline.getCudaPitchedPtr(), false );

    Spliner< Read1D<float2> > spliner(
            Read1D<float2>( spline.getNumberOfElements().x ),
            spline.getNumberOfElements().x,
            save_inside );

    elemSize3_t sz = data.getNumberOfElements();
    element_operate<float2>( data, make_float4(0, 0, sz.x, sz.y), spliner );

}
