#include "hip/hip_runtime.h"
#include "cudaUtil.h"
#include <stdio.h>
#include "wavelet.cu.h"

__global__ void kernel_compute( float* in_waveform_ft, float* out_wavelet_ft, hipExtent numElem, float start, float scales_per_octave, float steplogsize  );
__global__ void kernel_inverse( float2* in_wavelet, float* out_inverse_waveform, hipExtent numElem, unsigned n_valid_samples );
__global__ void kernel_inverse_ellips( float2* in_wavelet, float* out_inverse_waveform, hipExtent numElem, float4 area, unsigned n_valid_samples );
__global__ void kernel_inverse_box( float2* in_wavelet, float* out_inverse_waveform, hipExtent numElem, float4 area, unsigned n_valid_samples );
__global__ void kernel_clamp( float* in_wt, hipExtent in_numElem, size_t in_offset, size_t last_sample, float* out_clamped_wt, hipExtent out_numElem );

static const char* gLastError = 0;

const char* wtGetError() {
    const char* r = gLastError;
    gLastError = 0;
    return r;
}

void setError(const char* staticErrorMessage) {
    gLastError = staticErrorMessage;
    printf("%s\n", staticErrorMessage);
}

#define TOSTR(x) #x
#define setError(x) setError(TOSTR(__FUNCTION__) ": " x)

void wtCompute( float2* in_waveform_ft, float2* out_wavelet_ft, unsigned sampleRate, float minHz, float maxHz, hipExtent numElem, float scales_per_octave, hipStream_t stream )
{
    // in this scope, work on arrays of float* instead of float2* to coalesce better
    numElem.width *= 2;

    float start = sampleRate/minHz/numElem.width;
    float steplogsize = log(maxHz)-log(minHz);

    dim3 block(256,1,1);
    dim3 grid( int_div_ceil(numElem.width, block.x), numElem.depth, 1);

    if(grid.x>65535) {
        setError("Invalid argument, number of floats in complex signal must be less than 65535*256.");
        return;
    }

	// float scales_per_octave = numElem.height/((log(maxHz)/log(2.f)-(log(minHz)/log(2.f));
    kernel_compute<<<grid, block, 0, stream>>>( (float*)in_waveform_ft, (float*)out_wavelet_ft, numElem, start, steplogsize, scales_per_octave );
}

__global__ void kernel_compute(
        float* in_waveform_ft,
        float* out_wavelet_ft,
        hipExtent numElem, float start, float steplogsize, float scales_per_octave )
{
    // Element number
    const unsigned
            x = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;

    if (x>=numElem.width)
        return;

    float waveform = in_waveform_ft[x];

    float cufft_normalize = rsqrt((float)numElem.width);
    float jibberish_normalization =  26.0993675428f/scales_per_octave;

    // Find period for this thread
    unsigned nFrequencies = numElem.height;
    unsigned channel = blockIdx.y;
    unsigned n = numElem.width;
    for( unsigned fi = 0; fi<nFrequencies; fi++) {
        float ff = fi/(float)nFrequencies;
        float period = start*exp(-ff*steplogsize);

        // Find offset for this wavelet scale
        unsigned offset = fi*n + channel*n*nFrequencies;


        // Compute value of analytic FT of wavelet
        const float f0 = .6f + 40*ff*ff*ff;
        const float pi = 3.141592654f;
        const float two_pi_f0 = 2.0f * pi * f0;
        const float multiplier = 1.8827925275534296252520792527491f;

        period *= f0;

        unsigned y = x/2; // compute equal results for the complex and scalar part
        float factor = 4*pi*y*period-two_pi_f0;
        float basic = multiplier * exp(-0.5f*factor*factor);

        float m = jibberish_normalization*cufft_normalize*basic*f0;
        //float m = cufft_normalize*basic*f0;
        //float m = basic*f0;
        out_wavelet_ft[offset + x] = m * waveform;
    }
}

void wtInverse( float2* in_wavelet, float* out_inverse_waveform, hipExtent numElem, unsigned n_valid_samples, hipStream_t stream )
{
    // Multiply the coefficients together and normalize the result
    dim3 block(256,1,1);
    dim3 grid( int_div_ceil(numElem.width, block.x), 1, 1);

    if(grid.x>65535) {
        setError("Invalid argument, number of floats in complex signal must be less than 65535*256.");
        return;
    }

    kernel_inverse<<<grid, block, 0, stream>>>( in_wavelet, out_inverse_waveform, numElem, n_valid_samples );
}

__global__ void kernel_inverse( float2* in_wavelet, float* out_inverse_waveform, hipExtent numElem, unsigned n_valid_samples )
{
    const unsigned
            //x = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
            x = blockIdx.x*blockDim.x + threadIdx.x;

    if (x>=n_valid_samples)
        return;
    if (x>=numElem.width )
        return;

    float a = 0;

    // no selection
    for (unsigned fi=0; fi<numElem.height; fi++)
    {
        a += in_wavelet[ x + fi*numElem.width ].x;
    }

    float cufft_normalize = 1.f/sqrt((float)numElem.width);

    // TODO wtf?
    float jibberish_normalization = .1;

    out_inverse_waveform[x] = jibberish_normalization*cufft_normalize*a;
}

void wtInverseEllips( float2* in_wavelet, float* out_inverse_waveform, hipExtent numElem, float4 area, unsigned n_valid_samples, hipStream_t stream )
{
    // Multiply the coefficients together and normalize the result
    dim3 block(256,1,1);
    dim3 grid( int_div_ceil(numElem.width, block.x), 1, 1);

    if(grid.x>65535) {
        setError("Invalid argument, number of floats in complex signal must be less than 65535*256.");
        return;
    }

    kernel_inverse_ellips<<<grid, block, 0, stream>>>( in_wavelet, out_inverse_waveform, numElem, area, n_valid_samples );
}

__global__ void kernel_inverse_ellips( float2* in_wavelet, float* out_inverse_waveform, hipExtent numElem, float4 area, unsigned n_valid_samples )
{
    const unsigned
            //x = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
            x = blockIdx.x*blockDim.x + threadIdx.x;

    if (x>=n_valid_samples)
        return;
    if (x>=numElem.width )
        return;

    float a = 0;

    // disc selection
    for (unsigned fi=0; fi<numElem.height; fi++)
    {
        float rx = area.z-area.x;
        float ry = area.w-area.y;
        float dx = x-area.x;
        float dy = fi-area.y;

        if (dx*dx/rx/rx + dy*dy/ry/ry < 1) {
            // select only the real component of the complex transform
            a += in_wavelet[ x + fi*numElem.width ].x;
        }
    }

    float cufft_normalize = 1.f/sqrt((float)numElem.width);
    float jibberish_normalization = .1;

    out_inverse_waveform[x] = jibberish_normalization*cufft_normalize*a;
}

void wtInverseBox( float2* in_wavelet, float* out_inverse_waveform, hipExtent numElem, float4 area, unsigned n_valid_samples, hipStream_t stream )
{
    // Multiply the coefficients together and normalize the result
    dim3 block(256,1,1);
    dim3 grid( int_div_ceil(numElem.width, block.x), 1, 1);

    if(grid.x>65535) {
        setError("Invalid argument, number of floats in complex signal must be less than 65535*256.");
        return;
    }

    kernel_inverse_box<<<grid, block, 0, stream>>>( in_wavelet, out_inverse_waveform, numElem, area, n_valid_samples );
}

__global__ void kernel_inverse_box( float2* in_wavelet, float* out_inverse_waveform, hipExtent numElem, float4 area, unsigned n_valid_samples )
{
    const unsigned
            //x = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
            x = blockIdx.x*blockDim.x + threadIdx.x;

    if (x>=n_valid_samples)
        return;
    if (x>=numElem.width )
        return;

    float a = 0;

    // box selection
    if (x>=area.x && x<=area.z)
      {
        for (unsigned fi=max(0.f,area.y); fi<numElem.height && fi<area.w; fi++)
        {
            // select only the real component of the complex transform
            a += in_wavelet[ x + fi*numElem.width ].x;
        }
    }

    float cufft_normalize = 1.f/sqrt((float)numElem.width);
    float jibberish_normalization = .1;

    out_inverse_waveform[x] = jibberish_normalization*cufft_normalize*a;
}
