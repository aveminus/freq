#include "hip/hip_runtime.h"
#include "cudaUtil.h"
#include <stdio.h>

__global__ void kernel_compute( float* in_waveform_ft, float* out_wavelet_ft, hipExtent numElem, float start, float steplogsize  );
__global__ void kernel_inverse( float2* in_wavelet, float* out_inverse_waveform, hipExtent numElem );
__global__ void kernel_clamp( float* in_wt, hipExtent in_numElem, size_t in_offset, size_t last_sample, float* out_clamped_wt, hipExtent out_numElem );

static const char* gLastError = 0;

const char* wtGetError() {
    const char* r = gLastError;
    gLastError = 0;
    return r;
}

void setError(const char* staticErrorMessage) {
    gLastError = staticErrorMessage;
    printf("%s\n", staticErrorMessage);
}

#define TOSTR(x) #x
#define setError(x) setError(TOSTR(__FUNCTION__) ": " x)

void wtCompute( float2* in_waveform_ft, float2* out_wavelet_ft, unsigned sampleRate, float minHz, float maxHz, hipExtent numElem, hipStream_t stream )
{
    // in this scope, work on arrays of float* instead of float2* to coalesce better
    numElem.width *= 2;

    float start = sampleRate/minHz/numElem.width;
    float steplogsize = log(maxHz)-log(minHz);

    dim3 block(256,1,1);
    dim3 grid( INTDIV_CEIL(numElem.width, block.x), numElem.height*numElem.depth, 1);

    if(grid.x>65535) {
        setError("Invalid argument, number of floats in complex signal must be less than 65535*256.");
        return;
    }

    kernel_compute<<<grid, block, stream>>>( (float*)in_waveform_ft, (float*)out_wavelet_ft, numElem, start, steplogsize );
}

__global__ void kernel_compute(
        float* in_waveform_ft,
        float* out_wavelet_ft,
        hipExtent numElem, float start, float steplogsize )
{
    // Find period for this thread
    unsigned nFrequencies = numElem.height;
    unsigned fi = blockIdx.y%nFrequencies;
    float ff = fi/(float)nFrequencies;
    float period = start*exp(-ff*steplogsize);

    // Find offset for this wavelet scale
    unsigned channel = blockIdx.y/nFrequencies; // integer division
    unsigned n = numElem.width;
    unsigned offset = fi*n + channel*n*nFrequencies;

    // Element number
    const unsigned
            x = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;

    if (x>=numElem.width)
        return;

    // Compute value of analytic FT of wavelet
    const float f0 = .6f + 40*ff*ff*ff;
    const float pi = 3.141592654f;
    const float two_pi_f0 = 2.0f * pi * f0;
    const float multiplier = 1.8827925275534296252520792527491f;

    period *= f0;

    unsigned y = x/2; // compute equal results for the complex and scalar part
    float factor = 4*pi*y*period-two_pi_f0;
    float basic = multiplier * exp(-0.5f*factor*factor);

    float jibberish_normalization = 2.3406;
    float cufft_normalize = 1.f/numElem.width;

    out_wavelet_ft[offset + x] = jibberish_normalization*cufft_normalize*basic*f0*in_waveform_ft[x];
}

void wtInverse( float2* in_wavelet, float* out_inverse_waveform, hipExtent numElem, hipStream_t stream  )
{
    // Multiply the coefficients together and normalize the result
    dim3 block(256,1,1);
    dim3 grid( INTDIV_CEIL(numElem.width, block.x), 1, 1);

    if(grid.x>65535) {
        setError("Invalid argument, number of floats in complex signal must be less than 65535*256.");
        return;
    }

    kernel_inverse<<<grid, block, stream>>>( in_wavelet, out_inverse_waveform, numElem );
}

__global__ void kernel_inverse( float2* in_wavelet, float* out_inverse_waveform, hipExtent numElem )
{
    const unsigned
            x = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;

    if (x>=numElem.width )
        return;

    float a = 0;
    for (unsigned fi=0; fi<numElem.height; fi++)
    {
        a += in_wavelet[ x + fi*numElem.width ].x;
    }

    out_inverse_waveform[x] = a;
}

void wtClamp( float2* in_wt, hipExtent in_numElem, size_t in_offset, size_t last_sample, float2* out_clamped_wt, hipExtent out_numElem, hipStream_t stream )
{
    // in this scope, work on arrays of float* instead of float2* to coalesce better
    in_numElem.width *= 2;
    in_offset *= 2;
    out_numElem.width *= 2;
    last_sample *= 2;

    // Multiply the coefficients together and normalize the result
    dim3 block(256,1,1);
    dim3 grid( INTDIV_CEIL(out_numElem.width, block.x), out_numElem.height, out_numElem.depth );

    if(grid.x>65535) {
        setError("Invalid argument, first dimension of wavelet transform must be less than 65535*256 ~ 16 Mi.");
        return;
    }
    if(grid.y>65535) {
        setError("Invalid argument, number of scales in wavelet transform must be less than 65535.");
        return;
    }
    if(grid.z>1) {
        setError("Invalid argument, out_numElem.depth must be 1.");
        return;
    }

    kernel_clamp<<<grid, block, stream>>>( (float*)in_wt, in_numElem, in_offset, last_sample, (float*)out_clamped_wt, out_numElem );
}

__global__ void kernel_clamp( float* in_wt, hipExtent in_numElem, size_t in_offset, size_t last_sample, float* out_clamped_wt, hipExtent out_numElem )
{
    const unsigned
            x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x,
            y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;

    // sanity checks...
    if (x>=out_numElem.width )
        return;
    if (y>=out_numElem.height)
        return;

    // Not coalesced reads for arbitrary in_offset, coalesced writes though
    float v = 0;
    if (y<in_numElem.height && in_offset + x < in_numElem.width) {
        unsigned i = in_offset + x + in_numElem.width*y;
        v = in_wt[i];
    }

    if (x >= last_sample)
        v = 0.f/0.f;

    unsigned o = x + out_numElem.width*y;
    out_clamped_wt[o] = v;
}
