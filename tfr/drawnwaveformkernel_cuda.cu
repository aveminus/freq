#include "hip/hip_runtime.h"
#include "resamplecuda.cu.h"
#include "cuda_vector_types_op.h"
#include "drawnwaveformkerneldef.h"
#include <stdio.h>


template<typename Reader, typename Writer>
__global__ void kernel_draw_waveform(
        Reader in_waveform,
        Writer out_waveform_matrix, float blob, unsigned readstop, float scaling );


template<typename Reader, typename Writer>
__global__ void kernel_draw_waveform_with_lines(
        Reader in_waveform,
        Writer out_waveform_matrix, float blob, unsigned readstop, float scaling );


void drawWaveform(
        DataStorage<float>::Ptr in_waveformp,
        Tfr::ChunkData::Ptr out_waveform_matrixp,
        float blob, unsigned readstop, float maxValue )
{
    CudaGlobalReadOnly<float, 1> in_waveform = CudaGlobalStorage::ReadOnly<1>( in_waveformp );
    CudaGlobalReadWrite<float2, 2> out_waveform_matrix( CudaGlobalStorage::ReadWrite<2>( out_waveform_matrixp ).getCudaPitchedPtr() );

    unsigned w = out_waveform_matrixp->size().width;
    dim3 block(drawWaveform_BLOCK_SIZE, 1, 1);
    dim3 grid(int_div_ceil(w, block.x), 1, 1);

    if(grid.x>65535) {
        printf("Invalid argument, number of floats in complex signal must be less than 65535*256.");
        return;
    }

    if (blob > 1)
    {
        printf("blob > 1: %g", blob);
        kernel_draw_waveform<<<grid, block, 0, 0>>>( in_waveform, out_waveform_matrix, blob, readstop, 1.f/maxValue );
    }
    else
    {
        printf("blob <= 1: %g", blob);
        kernel_draw_waveform_with_lines<<<grid, block, 0, 0>>>( in_waveform, out_waveform_matrix, blob, readstop, 1.f/maxValue );
    }
}


template<typename Reader, typename Writer>
__global__ void kernel_draw_waveform(
        Reader in_waveform,
        Writer out_waveform_matrix, float blob, unsigned readstop, float scaling )
{
    unsigned writePos_x = blockIdx.x * blockDim.x + threadIdx.x;

    draw_waveform_elem(
            writePos_x,
            in_waveform,
            out_waveform_matrix, blob, readstop, scaling );
}


template<typename Reader, typename Writer>
__global__ void kernel_draw_waveform_with_lines(
        Reader in_waveform,
        Writer out_waveform_matrix, float blob, unsigned readstop, float scaling )
{
    unsigned writePos_x = blockIdx.x * blockDim.x + threadIdx.x;

    draw_waveform_with_lines_elem(
            writePos_x,
            in_waveform,
            out_waveform_matrix, blob, readstop, scaling );
}
