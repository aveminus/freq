#include "hip/hip_runtime.h"
#include "drawnwaveform.cu.h"
#include <stdio.h>

#include "cudaglobalstorage.h"

/**
 Plot the waveform on the matrix.

 Not coalesced, could probably be optimized.
 */
__global__ void kernel_draw_waveform(
        cudaPitchedPtrType<float> in_waveform,
        cudaPitchedPtrType<float2> out_waveform_matrix,
        float blob, unsigned readstop, float scaling );


__global__ void kernel_draw_waveform_with_lines(
        cudaPitchedPtrType<float> in_waveform,
        cudaPitchedPtrType<float2> out_waveform_matrix,
        float blob, unsigned readstop, float scaling );


void drawWaveform(
        DataStorage<float>::Ptr in_waveformp,
        Tfr::ChunkData::Ptr out_waveform_matrixp,
        float blob, unsigned readstop, float maxValue )
{
    cudaPitchedPtrType<float> in_waveform(CudaGlobalStorage::ReadOnly<1>( in_waveformp ).getCudaPitchedPtr());

    cudaPitchedPtrType<float2> out_waveform_matrix(CudaGlobalStorage::ReadWrite<2>( in_waveformp ).getCudaPitchedPtr());

    hipMemset( out_waveform_matrix.ptr(), 0, out_waveform_matrix.getTotalBytes() );

    unsigned w = out_waveform_matrix.getNumberOfElements().x;
    dim3 block(drawWaveform_BLOCK_SIZE, 1, 1);
    dim3 grid(int_div_ceil(w, block.x), 1, 1);

    if(grid.x>65535) {
        printf("Invalid argument, number of floats in complex signal must be less than 65535*256.");
        return;
    }

    if (blob > 1)
    {
        printf("blob > 1: %g", blob);
        kernel_draw_waveform<<<grid, block, 0, 0>>>( in_waveform, out_waveform_matrix, blob, readstop, 1.f/maxValue );
    }
    else
    {
        printf("blob <= 1: %g", blob);
        kernel_draw_waveform_with_lines<<<grid, block, 0, 0>>>( in_waveform, out_waveform_matrix, blob, readstop, 1.f/maxValue );
    }
}


__global__ void kernel_draw_waveform(
        cudaPitchedPtrType<float> in_waveform,
        cudaPitchedPtrType<float2> out_waveform_matrix, float blob, unsigned readstop, float scaling )
{
    elemSize_t writePos_x = blockIdx.x * blockDim.x + threadIdx.x;
    elemSize3_t matrix_sz = out_waveform_matrix.getNumberOfElements();
    elemSize_t readPos1 = writePos_x * blob;
    elemSize_t readPos2 = (writePos_x + 1) * blob;
    elemSize3_t writePos;

    if( writePos_x >= matrix_sz.x || readPos1 >= readstop )
        return;

    float blobinv = 1.f/blob;

    for (elemSize_t read_x = readPos1; read_x<readPos2 && read_x < readstop; ++read_x)
    {
        elemSize3_t readPos = make_elemSize3_t(read_x, 0, 0);

        float v = in_waveform.elem( readPos );
        v *= scaling;
        v = fmaxf(-1.f, fminf(1.f, v));
        float y = (v+1.f)*.5f*(matrix_sz.y-1.f);
        elemSize_t y1 = (elemSize_t)y;
        elemSize_t y2 = y1+1;
        if (y2 >= matrix_sz.y)
        {
            y2 = matrix_sz.y - 1;
            y1 = y2 - 1;
        }
        float py = y-y1;

        writePos = make_elemSize3_t( writePos_x, y1, 0 );
        out_waveform_matrix.e( writePos ).x += 0.8f*blobinv * (1.f-py);

        writePos = make_elemSize3_t( writePos_x, y2, 0 );
        out_waveform_matrix.e( writePos ).x += 0.8f*blobinv * py;
    }
}


__global__ void kernel_draw_waveform_with_lines(
        cudaPitchedPtrType<float> in_waveform,
        cudaPitchedPtrType<float2> out_waveform_matrix, float blob, unsigned readstop, float scaling )
{
    elemSize_t writePos_x = blockIdx.x * blockDim.x + threadIdx.x;
    elemSize3_t matrix_sz = out_waveform_matrix.getNumberOfElements();
    elemSize_t readPos = writePos_x * blob;
    float px = writePos_x * blob - readPos;

    if( writePos_x >= matrix_sz.x || readPos >= readstop )
        return;

    float blobinv = 1.f/blob;

    float v1 = in_waveform.elem( make_elemSize3_t(readPos, 0, 0) );
    float v2 = in_waveform.elem( make_elemSize3_t(readPos+1, 0, 0) );
    float v = v1*(1-px) + v2*px;
    v *= scaling;
    v = fmaxf(-1.f, fminf(1.f, v));
    float y = (v+1.f)*.5f*(matrix_sz.y-1.f);
    elemSize_t y1 = (elemSize_t)y;
    elemSize_t y2 = y1+1;
    if (y2 >= matrix_sz.y)
    {
        y2 = matrix_sz.y - 1;
        y1 = y2 - 1;
    }
    float py = y-y1;

    elemSize3_t writePos = make_elemSize3_t( writePos_x, y1, 0 );
    out_waveform_matrix.e( writePos ).x += 0.8f*blobinv * (1.f-py);

    writePos = make_elemSize3_t( writePos_x, y2, 0 );
    out_waveform_matrix.e( writePos ).x += 0.8f*blobinv * py;
}
