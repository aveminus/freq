#include "hip/hip_runtime.h"
#ifndef WAVELETKERNEL_CUDA_CU
#define WAVELETKERNEL_CUDA_CU

#include <stdio.h>

#include "resamplecuda.cu.h"
#include "cuda_vector_types_op.h"
#include "waveletkerneldef.h"

__global__ void kernel_compute_wavelet_coefficients( float2* in_waveform_ft, float2* out_wavelet_ft, unsigned nFrequencyBins, unsigned nScales, float first_j, float v, unsigned half_sizes, float sigma_t0, float normalization_factor );
__global__ void kernel_inverse( float2* in_wavelet, float* out_inverse_waveform, hipExtent numElem );
//__global__ void kernel_inverse_ellipse( float2* in_wavelet, float* out_inverse_waveform, hipExtent numElem, float4 area, unsigned n_valid_samples );
//__global__ void kernel_inverse_box( float2* in_wavelet, float* out_inverse_waveform, hipExtent numElem, float4 area, unsigned n_valid_samples );
__global__ void kernel_clamp( cudaPitchedPtrType<float2> in_wt, size_t sample_offset, cudaPitchedPtrType<float2> out_clamped_wt );
__global__ void kernel_stftNormalizeInverse( cudaPitchedPtrType<float> wave, float v );

static const char* gLastError = 0;

const char* wtGetError() {
    const char* r = gLastError;
    gLastError = 0;
    return r;
}

void setError(const char* staticErrorMessage) {
    gLastError = staticErrorMessage;
    printf("%s\n", staticErrorMessage);
}

#define TOSTR2(x) #x
#define TOSTR(x) TOSTR2(x)

#ifdef _MSC_VER
    #define setError(x) setError(__FUNCTION__ ": " x)
#endif

#if 0
void wtCompute(
        DataStorage<Tfr::ChunkElement>::Ptr in_waveform_ftp,
        Tfr::ChunkData::Ptr out_wavelet_ftp,
        float fs,
        float /*minHz*/,
        float maxHz,
        unsigned half_sizes,
        float scales_per_octave,
        float sigma_t0,
        float normalization_factor )
{
    CudaGlobalStorage::useCudaPitch( out_wavelet_ftp, false );

    Tfr::ChunkElement* in_waveform_ft = CudaGlobalStorage::ReadOnly<1>( in_waveform_ftp ).device_ptr();
    Tfr::ChunkElement* out_wavelet_ft = CudaGlobalStorage::WriteAll<2>( out_wavelet_ftp ).device_ptr();

    DataStorageSize size = out_wavelet_ft->size();

//    nyquist = FS/2
//    a = 2 ^ (1/v)
//    aj = a^j
//    hz = fs/2/aj
//    maxHz = fs/2/(a^j)
//    (a^j) = fs/2/maxHz
//    exp(log(a)*j) = fs/2/maxHz
//    j = log(fs/2/maxHz) / log(a)
//    const float log2_a = log2f(2.f) / v = 1.f/v; // a = 2^(1/v)
    float j = (log2f(fs/2) - log2f(maxHz)) * scales_per_octave;
    float first_scale = j;

    j = floor(j+0.5f);

    if (j<0) {
        printf("j = %g, maxHz = %g, fs = %g\n", j, maxHz, fs);
        setError("Invalid argument, maxHz must be less than or equal to fs/2.");
        return;
    }

    dim3 block(256,1,1);
    dim3 grid( int_div_ceil(size.width, block.x), 1, 1);

    if(grid.x>65535) {
        setError("Invalid argument, number of floats in complex signal must be less than 65535*256.");
        return;
    }

    kernel_compute_wavelet_coefficients<<<grid, block, 0>>>(
            (float2*)in_waveform_ft,
            (float2*)out_wavelet_ft,
            size.width, size.height,
            first_scale,
            scales_per_octave,
            half_sizes,
            sigma_t0,
            normalization_factor );
}
#endif

__global__ void kernel_compute_wavelet_coefficients(
        float2* in_waveform_ft,
        float2* out_wavelet_ft,
        unsigned nFrequencyBins, unsigned nScales, float first_scale, float v, unsigned half_sizes, float sigma_t0,
        float normalization_factor )
{
    // Which frequency bin in the discrete fourier transform this thread
    // should work with
    const unsigned
            w_bin = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;

    compute_wavelet_coefficients_elem(
            w_bin,
            in_waveform_ft,
            out_wavelet_ft,
            nFrequencyBins,
            nScales,
            first_scale,
            v,
            half_sizes,
            sigma_t0,
            normalization_factor);
}

#if 0
void wtInverse( Tfr::ChunkData::Ptr in_waveletp, DataStorage<float>::Ptr out_inverse_waveform, DataStorageSize x )
{
    // Multiply the coefficients together and normalize the result
    dim3 block(256,1,1);
    dim3 grid( int_div_ceil(x.width, block.x), 1, 1);

    if(grid.x>65535) {
        setError("Invalid argument, number of floats in complex signal must be less than 65535*256.");
        return;
    }

    // kernel_inverse<<<grid, block, 0, stream>>>( in_wavelet, out_inverse_waveform, numElem );
    kernel_inverse<<<grid, block>>>(
            (float2*)CudaGlobalStorage::ReadOnly<2>(in_waveletp).device_ptr(),
            CudaGlobalStorage::WriteAll<1>(out_inverse_waveform).device_ptr(),
            x );
}
#endif

__global__ void kernel_inverse( float2* in_wavelet, float* out_inverse_waveform, DataStorageSize numElem )
{
    const unsigned
            x = blockIdx.x*blockDim.x + threadIdx.x;

    inverse_elem( x, in_wavelet, out_inverse_waveform, numElem );
}


/*
void wtInverseEllipse( float2* in_wavelet, float* out_inverse_waveform, hipExtent numElem, float4 area, unsigned n_valid_samples, hipStream_t stream )
{
    // Multiply the coefficients together and normalize the result
    dim3 block(256,1,1);
    dim3 grid( int_div_ceil(numElem.width, block.x), 1, 1);

    if(grid.x>65535) {
        setError("Invalid argument, number of floats in complex signal must be less than 65535*256.");
        return;
    }

    kernel_inverse_ellipse<<<grid, block, 0, stream>>>( in_wavelet, out_inverse_waveform, numElem, area, n_valid_samples );
}

__global__ void kernel_inverse_ellipse( float2* in_wavelet, float* out_inverse_waveform, hipExtent numElem, float4 area, unsigned n_valid_samples )
{
    const unsigned
            x = blockIdx.x*blockDim.x + threadIdx.x;

    if (x>=n_valid_samples)
        return;
    if (x>=numElem.width )
        return;

    float a = 0;

    // disc selection
    for (unsigned fi=0; fi<numElem.height; fi++)
    {
        float rx = area.z-area.x;
        float ry = area.w-area.y;
        float dx = x-area.x;
        float dy = fi-area.y;

        if (dx*dx/rx/rx + dy*dy/ry/ry < 1) {
            // select only the real component of the complex transform
            a += in_wavelet[ x + fi*numElem.width ].x;
        }
    }

    out_inverse_waveform[x] = a;
}

void wtInverseBox( float2* in_wavelet, float* out_inverse_waveform, hipExtent numElem, float4 area, unsigned n_valid_samples, hipStream_t stream )
{
    // Multiply the coefficients together and normalize the result
    dim3 block(256,1,1);
    dim3 grid( int_div_ceil(numElem.width, block.x), 1, 1);

    if(grid.x>65535) {
        setError("Invalid argument, number of floats in complex signal must be less than 65535*256.");
        return;
    }

    kernel_inverse_box<<<grid, block, 0, stream>>>( in_wavelet, out_inverse_waveform, numElem, area, n_valid_samples );
}

__global__ void kernel_inverse_box( float2* in_wavelet, float* out_inverse_waveform, hipExtent numElem, float4 area, unsigned n_valid_samples )
{
    const unsigned
            x = blockIdx.x*blockDim.x + threadIdx.x;

    if (x>=n_valid_samples)
        return;
    if (x>=numElem.width )
        return;

    float a = 0;

    // box selection
    if (x>=area.x && x<=area.z)
      {
        for (unsigned fi=max(0.f,area.y); fi<numElem.height && fi<area.w; fi++)
        {
            float2 v = in_wavelet[ x + fi*numElem.width ];
            // select only the real component of the complex transform
            a += v.x;
        }
    }

    out_inverse_waveform[x] = a;
}
*/
void wtClamp( Tfr::ChunkData::Ptr in_wtp, size_t sample_offset, Tfr::ChunkData::Ptr out_clamped_wtp )
{
    cudaPitchedPtrType<float2> in_wt(CudaGlobalStorage::ReadOnly<2>( in_wtp ).getCudaPitchedPtr());
    cudaPitchedPtrType<float2> out_clamped_wt(CudaGlobalStorage::WriteAll<2>( out_clamped_wtp ).getCudaPitchedPtr());
    // Multiply the coefficients together and normalize the result

    dim3 grid, block;
    unsigned block_size = 256;
    out_clamped_wt.wrapCudaGrid2D( block_size, grid, block );

    if(grid.x>65535) {
        setError("Invalid argument, number of floats in complex signal must be less than 65535*256.");
        return;
    }

    kernel_clamp<<<grid, block, 0>>>( in_wt, sample_offset, out_clamped_wt );
}

__global__ void kernel_clamp( cudaPitchedPtrType<float2> in_wt, size_t sample_offset, cudaPitchedPtrType<float2> out_clamped_wt )
{
    elemSize3_t writePos;
    if( !out_clamped_wt.unwrapCudaGrid( writePos ))
        return;

    elemSize3_t readPos = writePos;
    readPos.x += sample_offset;

    out_clamped_wt.e( writePos ) = in_wt.elem(readPos);
}

void stftNormalizeInverse(
        DataStorage<float>::Ptr wavep,
        unsigned length )
{
    // Multiply the coefficients together and normalize the result
    cudaPitchedPtrType<float> wave(CudaGlobalStorage::ReadWrite<1>( wavep ).getCudaPitchedPtr());

    dim3 grid, block;
    unsigned block_size = 256;
    wave.wrapCudaGrid2D( block_size, grid, block );

    if(grid.x>65535) {
        setError("Invalid argument, number of floats in complex signal must be less than 65535*256.");
        return;
    }

    kernel_stftNormalizeInverse<<<grid, block, 0>>>( wave, 1.f/length );
}


__global__ void kernel_stftNormalizeInverse( cudaPitchedPtrType<float> wave, float v )
{
    elemSize3_t writePos;
    if( !wave.unwrapCudaGrid( writePos ))
        return;

    wave.e( writePos ) *= v;
}

#endif // WAVELETKERNEL_CUDA_CU
