#include "hip/hip_runtime.h"
#include "resampletest.cu.h"
#include <stdio.h>

#include <resamplecuda.cu.h>

void simple_resample2d_2(
        cudaPitchedPtrType<float2> input,
        cudaPitchedPtrType<float> output
        )
{
    void (*myptr)(cudaPitchedPtrType<float2>,cudaPitchedPtrType<float>);
    myptr = &simple_resample2d_2;
    printf("&simple_resample2d_2 = %p\n", (void*)myptr);

    resample2d_plain<ConverterAmplitude >(
            input,
            output,
            ResampleArea(0.1, 0.1, 0.9, 0.9),
//            make_float4(0.0,0.0,0.5,0.5),
//            make_float4(0.0,0.0,0.5,0.5),
//            make_float4(0.0,0.0,0.5,0.5),
            ResampleArea(0, 0, 1, 1),
//            make_float4(0,0,1,1),
            false
    );
}
