#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include <GL/glut.h>
#include <iostream>
#include <fstream>
#include <cuda_gl_interop.h>

using namespace std;

#define BLOCK_SIZE 128
__global__ void simpleKernel(
        float* output )
{
    output[threadIdx.x] = 0;
}

hipGraphicsResource* positionsVBO_CUDA;

void display()
{
    glClear(GL_COLOR_BUFFER_BIT);
	glutSwapBuffers();
	
    static bool once = true;
	if (!once)
	    return;
    once = false;
	
    hipError_t cuda_inited = cudaGLSetGLDevice(0);
    int glew_inited = glewInit();

    unsigned N = BLOCK_SIZE;
    unsigned size = N*sizeof(float);
    unsigned vbo;
    glGenBuffers(1, &vbo);
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    hipError_t is_registered = hipGraphicsGLRegisterBuffer( &positionsVBO_CUDA, vbo, cudaGraphicsMapFlagsWriteDiscard);
    hipError_t is_mapped = hipGraphicsMapResources(1, &positionsVBO_CUDA, 0 );
    float* g_data;
    size_t num_bytes;
    hipError_t got_pointer = hipGraphicsResourceGetMappedPointer((void**)&g_data, &num_bytes, positionsVBO_CUDA);

    float* g_data2;
    hipError_t mallocd = hipMalloc( &g_data2, size );
    hipError_t memcopied = hipMemcpy( g_data2, g_data, size, hipMemcpyDeviceToDevice );

    dim3 block( BLOCK_SIZE );
    dim3 grid( 1 );
    simpleKernel<<< grid, block>>>(g_data2);
    hipError_t freed = hipFree( g_data2 );

    hipError_t unmapped = hipGraphicsUnmapResources(1, &positionsVBO_CUDA, 0);
    hipError_t unreg = hipGraphicsUnregisterResource( positionsVBO_CUDA );
	hipError_t sync = hipDeviceSynchronize();

    cout << "cuda_inited = " << (cuda_inited == hipSuccess) << endl
        << "glew_inited = " << (glew_inited == 0) << endl
        << "is_registered = "<< (is_registered == hipSuccess) << endl
        << "is_mapped = "<< (is_mapped == hipSuccess) << endl
        << "num_bytes = " << num_bytes << endl
        << "g_data = " << g_data << endl
        << "got_pointer = " << (got_pointer == hipSuccess) << endl
		<< "mallocd = " << (mallocd == hipSuccess) << endl
		<< "memcopied = " << (memcopied == hipSuccess) << endl
		<< "freed = " << (freed == hipSuccess) << endl
        << "unmapped = "<< (unmapped == hipSuccess) << endl
        << "unreg = "<< (unreg == hipSuccess) << endl
        << "sync = "<< (sync == hipSuccess) << endl;
	
    string name = __FILE__ " log.txt";

    ofstream tst(name.c_str());
    tst << name.c_str() << endl
        << "cuda_inited = " << (cuda_inited == hipSuccess) << endl
        << "glew_inited = " << (glew_inited == 0) << endl
        << "is_registered = "<< (is_registered == hipSuccess) << endl
        << "is_mapped = "<< (is_mapped == hipSuccess) << endl
        << "num_bytes = " << num_bytes << endl
        << "g_data = " << g_data << endl
        << "got_pointer = "<< (got_pointer == hipSuccess) << endl
		<< "mallocd = " << (mallocd == hipSuccess) << endl
		<< "memcopied = " << (memcopied == hipSuccess) << endl
		<< "freed = " << (freed == hipSuccess) << endl
        << "unmapped = "<< (unmapped == hipSuccess) << endl
        << "unreg = "<< (unreg == hipSuccess) << endl
        << "sync = "<< (sync == hipSuccess) << endl;
}


int main(int argc, char *argv[])
{
    glutInit(&argc,argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_DOUBLE | GLUT_DEPTH);
    glutInitWindowSize(500, 500);
    glutInitWindowPosition(300, 200);
    glutCreateWindow(__FILE__);
    glutDisplayFunc( display );
    glutMainLoop();
    return 0;
}
