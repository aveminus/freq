#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>

using namespace std;

#define BLOCK_SIZE 128
__global__ void simpleKernel(
        float* output )
{
    output[threadIdx.x] = 0;
}


int main(int argc, char *argv[])
{
    unsigned N = BLOCK_SIZE;
    unsigned size = N*sizeof(float);
    float* g_data;
    hipError_t mallocd = hipMalloc( &g_data, size );
    dim3 block( BLOCK_SIZE );
    dim3 grid( 1 );
    simpleKernel<<< grid, block>>>(g_data);
    hipError_t freed = hipFree( g_data );
	
	hipError_t sync = hipDeviceSynchronize();

    cout << "mallocd = " << (mallocd == hipSuccess) << endl
         << "freed = " << (freed == hipSuccess) << endl
         << "sync = " << (sync == hipSuccess) << endl;
	
    string name = __FILE__ " log.txt";

    ofstream tst(name.c_str());
    tst  << name.c_str() << endl
         << "mallocd = " << (mallocd == hipSuccess) << endl
         << "freed = " << (freed == hipSuccess) << endl
         << "sync = " << (sync == hipSuccess) << endl;
}
