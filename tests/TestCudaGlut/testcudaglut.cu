#include "hip/hip_runtime.h"
#include <GL/glut.h>
#include <iostream>
#include <fstream>

using namespace std;

#define BLOCK_SIZE 128
__global__ void simpleKernel(
        float* output )
{
    output[threadIdx.x] = 0;
}

hipGraphicsResource* positionsVBO_CUDA;

void display()
{
    glClear(GL_COLOR_BUFFER_BIT);
	glutSwapBuffers();
	
    static bool once = true;
	if (!once)
	    return;
    once = false;
	
    unsigned N = BLOCK_SIZE;
    unsigned size = N*sizeof(float);
    float* g_data;
    hipError_t mallocd = hipMalloc( &g_data, size );
    dim3 block( BLOCK_SIZE );
    dim3 grid( 1 );
    simpleKernel<<< grid, block>>>(g_data);
	
    hipError_t freed = hipFree( g_data );
	hipError_t sync = hipDeviceSynchronize();

    cout << "mallocd = " << (mallocd == hipSuccess) << endl
         << "freed = " << (freed == hipSuccess) << endl
         << "sync = " << (sync == 0) << endl;
	
    string name = __FILE__ " log.txt";

    ofstream tst(name.c_str());
    tst << name.c_str() << endl
         << "mallocd = " << (mallocd == hipSuccess) << endl
         << "freed = " << (freed == hipSuccess) << endl
         << "sync = " << (sync == 0) << endl;
}


int main(int argc, char *argv[])
{
    glutInit(&argc,argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_DOUBLE | GLUT_DEPTH);
    glutInitWindowSize(500, 500);
    glutInitWindowPosition(300, 200);
    glutCreateWindow(__FILE__);
    glutDisplayFunc( display );
    glutMainLoop();
    return 0;
}
