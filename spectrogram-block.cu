#include "hip/hip_runtime.h"
#include <stdio.h>
#include "spectrogram-block.cu.h"

__global__ void kernel_merge(
                cudaPitchedPtrType<float> inBlock,
                cudaPitchedPtrType<float> outBlock,
                float resample_width,
                float resample_height,
                float in_offset,
                float out_offset,
                float in_valid_samples)
{
    elemSize3_t writePos;
    if( !outBlock.unwrapCudaGrid( writePos ))
        return;

    float val = 0;
    //unsigned n = 0;

    if (writePos.x>=out_offset)
    {
        for (float x = 0; x < resample_width; x++)
        {
            float s = in_offset + x + resample_width*(writePos.x-out_offset);
            if ( s >= in_offset + in_valid_samples )
                continue;

            for (float y = 0; y < resample_height; y++)
            {
                float t = y + resample_height*writePos.y;

                elemSize3_t readPos = make_elemSize3_t( s, t, 0 );
                if ( inBlock.valid(readPos) ) {
                    val += inBlock.elem(readPos);

                    outBlock.e( writePos ) = val;
                    return;

                    //n ++;
                }
            }
        }
    }
/*
    if (0<n) {
        val/=n;
        outBlock.elem( writePos ) = val;
    }*/
}

extern "C"
void blockMerge( cudaPitchedPtrType<float> inBlock,
                 cudaPitchedPtrType<float> outBlock,
                 float in_sample_rate,
                 float out_sample_rate,
                 float in_frequency_resolution,
                 float out_frequency_resolution,
                 float in_offset,
                 float out_offset,
                 float in_valid_samples,
                 unsigned hip_stream)
{
    dim3 grid, block;
    unsigned block_size = 128;

    outBlock.wrapCudaGrid2D( block_size, grid, block );

    float resample_width = in_sample_rate/out_sample_rate;
    float resample_height = in_frequency_resolution/out_frequency_resolution;

    kernel_merge<<<grid, block, hip_stream>>>(
        inBlock, outBlock,
        resample_width,
        resample_height,
        in_offset, out_offset, in_valid_samples );
}

__global__ void kernel_merge_chunk(
                cudaPitchedPtrType<float2> inChunk,
                cudaPitchedPtrType<float> outBlock,
                float resample_width,
                float resample_height,
                float in_offset,
                float out_offset,
                unsigned n_valid_samples )
{
    elemSize3_t writePos;
    if( !outBlock.unwrapCudaGrid( writePos ))
        return;

    float val = 0;
    //unsigned n = 0;

    if (writePos.x>=out_offset)
    {
        for (float x = 0; x < resample_width; x++)
        {
            float s = in_offset + x + resample_width*(writePos.x-out_offset);
            if ( s >= in_offset + n_valid_samples )
                continue;

            for (float y = 0; y < resample_height; y++)
            {
                float t = y + resample_height*writePos.y;

                elemSize3_t readPos = make_elemSize3_t( s, t, 0 );
                if ( inChunk.valid(readPos) ) {
                    float2 c = inChunk.elem(readPos);
                    val += sqrt(c.x*c.x + c.y*c.y);

 outBlock.e( writePos ) = val;
 return;
/*
  TODO use command line argument "yscale"
                        case Yscale_Linear:
                            v[2][df] = amplitude;
                            break;
                        case Yscale_ExpLinear:
                            v[2][df] = amplitude * exp(.001*fi);
                            break;
                        case Yscale_LogLinear:
                            v[2][df] = amplitude;
                            v[2][df] = log(1+fabsf(v[2][df]))*(v[2][df]>0?1:-1);
                            break;
                        case Yscale_LogExpLinear:
                            v[2][df] = amplitude * exp(.001*fi);
                            v[2][df] = log(1+fabsf(v[2][df]))*(v[2][df]>0?1:-1);
                            */

  /*                  n ++;*/
                }
            }
        }
    }
/*
    __syncthreads();

    if (0<n) {
        val/=n;
        outBlock.e( writePos ) = val;
    }*/
}

/*
#define WARP 32

__global__ void kernel_merge_chunk(
                cudaPitchedPtrType<float2> inChunk,
                cudaPitchedPtrType<float> outBlock,
                float resample_width,
                float resample_height,
                float in_offset,
                float out_offset)
{
    elemSize3_t writePos;
    if( !outBlock.unwrapCudaGrid( writePos ))
        return;

    __shared__ float val[WARP] = 0;
    unsigned n = 0;

    if (writePos.x>=out_offset)
    {
        for (float x = 0; x < resample_width; x++)
        {
            for (float y = 0; y < resample_height; y++)
            {
                float s = in_offset + x + resample_width*(writePos.x-out_offset);
                float t = y + resample_height*writePos.y;

                elemSize3_t readPos = make_elemSize3_t( s, t, 0 );
                if ( inChunk.valid(readPos) ) {
                    unsigned o = inChunk.eOffs(readPos);
                    float* i = (float*)inChunk.ptr();
                    i[2*o + WARP%2];
                    float2 c
                    //val = max(val, sqrt(c.x*c.x + c.y*c.y)); n=0;
                    val += sqrt(c.x*c.x + c.y*c.y);
                    //val += c.x;

                    //outBlock.e( writePos ) = val;
                    //return;

                    n ++;
                }
            }
        }
    }

    __syncthreads();

    if (0<n && threadIdx.x < WARP) {
        val/=n;
        outBlock.e( writePos ) = val;
    }
}*/

extern "C"
void blockMergeChunk( cudaPitchedPtrType<float2> inChunk,
                 cudaPitchedPtrType<float> outBlock,
                 float in_sample_rate,
                 float out_sample_rate,
                 float in_frequency_resolution,
                 float out_frequency_resolution,
                 float in_offset,
                 float out_offset,
                 unsigned n_valid_samples,
                 unsigned hip_stream)
{
    dim3 grid, block;
    unsigned block_size = 128;

    outBlock.wrapCudaGrid2D( block_size, grid, block );

    float resample_width = in_sample_rate/out_sample_rate;
    float resample_height = in_frequency_resolution/out_frequency_resolution;

    if(0) {
        elemSize3_t sz_o = outBlock.getNumberOfElements();
        elemSize3_t sz_i = inChunk.getNumberOfElements();
        fprintf(stdout,"sz_o (%d, %d, %d)\tsz_i (%d, %d, %d)\n", sz_o.x, sz_o.y, sz_o.z, sz_i.x, sz_i.y, sz_i.z );


        fprintf(stdout,"grid (%d, %d, %d)\tblock (%d, %d, %d)\n", grid.x, grid.y, grid.z, block.x, block.y, block.z );
        fprintf(stdout,"in sr %g, out sr %g, in f %g, out f %g, in o %g, out o %g\n\tw=%g, h=%g\n",
            in_sample_rate, out_sample_rate,
            in_frequency_resolution, out_frequency_resolution,
            in_offset, out_offset,
            resample_width, resample_height);
        fprintf(stdout,"outBlock(%d,%d,%d) pitch %lu\n",
            outBlock.getNumberOfElements().x,
            outBlock.getNumberOfElements().y,
            outBlock.getNumberOfElements().z,
            outBlock.getCudaPitchedPtr().pitch );
        fprintf(stdout,"inChunk(%d,%d,%d) pitch %lu\n",
            inChunk.getNumberOfElements().x,
            inChunk.getNumberOfElements().y,
            inChunk.getNumberOfElements().z,
            inChunk.getCudaPitchedPtr().pitch );
        fflush(stdout);

    }

    kernel_merge_chunk<<<grid, block, hip_stream>>>(
        inChunk, outBlock,
        resample_width,
        resample_height,
        in_offset, out_offset, n_valid_samples );
}

__global__ void kernel_expand_stft(
                cudaPitchedPtrType<float2> inStft,
                cudaPitchedPtrType<float> outBlock,
                float start,
                float steplogsize,
                float out_offset,
                float out_length )
{
    // Element number
    const unsigned
            y = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;

    unsigned nFrequencies = outBlock.getNumberOfElementsD().y;
    if( y >= nFrequencies )
        return;

    float ff = y/(float)nFrequencies;
    float hz_out = start*exp(ff*steplogsize);

    float max_stft_hz = inStft.getNumberOfElementsD().x/2;
    float read_f = hz_out/max_stft_hz;

    float2 c;

    float p = read_f*inStft.getNumberOfElementsD().x;
    elemSize3_t readPos = make_elemSize3_t( p, 0, 0 );
    inStft.clamp(readPos);
    c = inStft.elem(readPos);
    float val1 = sqrt(c.x*c.x + c.y*c.y);

    readPos.x++;
    inStft.clamp(readPos);
    c = inStft.elem(readPos);
    float val2 = sqrt(c.x*c.x + c.y*c.y);

    p-=(unsigned)p;
    float val = val1*(1-p)+val2*p;

    elemSize3_t writePos = make_elemSize3_t( 0, y, 0 );
    for (writePos.x=out_offset; writePos.x<out_offset + out_length && writePos.x<outBlock.getNumberOfElementsD().x;writePos.x++)
    {
        outBlock.e( writePos ) = val;
    }
}

extern "C"
void expandStft( cudaPitchedPtrType<float2> inStft,
                 cudaPitchedPtrType<float> outBlock,
                 float min_hz,
                 float max_hz,
                 float out_offset,
                 float out_length,
                 unsigned hip_stream)
{
    dim3 block(256,1,1);
    dim3 grid( INTDIV_CEIL(outBlock.getNumberOfElements().x, block.y), 1, 1);

    if(grid.x>65535) {
        printf("====================\nInvalid argument, number of floats in complex signal must be less than 65535*256.\n===================\n");
        return;
    }

    float start = min_hz*outBlock.getNumberOfElements().x;
    float steplogsize = log(max_hz)-log(min_hz);

    kernel_expand_stft<<<grid, block, hip_stream>>>(
        inStft, outBlock,
        start,
        steplogsize,
        out_offset,
        out_length );
}
