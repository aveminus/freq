#include "hip/hip_runtime.h"
#include "rectangle.cu.h"

// gpumisc
#include <cudaUtil.h>

// stdc
#include <stdio.h>

__global__ void kernel_remove_rect(float2* in_wavelet, hipExtent in_numElem, float4 area, float save_inside );


void removeRect( float2* wavelet, hipExtent numElem, float4 area, bool save_inside )
{
    dim3 block(256,1,1);
    dim3 grid( int_div_ceil(numElem.width, block.x), numElem.height, 1);

    if(grid.x>65535) {
        printf("Invalid argument, number of floats in complex signal must be less than 65535*256.");
        return;
    }

    kernel_remove_rect<<<grid, block>>>( wavelet, numElem, area, save_inside );
}

__global__ void kernel_remove_rect(float2* wavelet, hipExtent numElem, float4 area, float save_inside )
{
    const unsigned
            x = blockIdx.x*blockDim.x + threadIdx.x,
            fi = blockIdx.y*blockDim.y + threadIdx.y;

    if (x>=numElem.width )
        return;
    /*float dx = area.x;
    float dy = area.y;
    float dh = area.z - area.x;
    float dw = area.w - area.y;*/
    float f;

    //if(x > dx - dh && x < dx + dh && fi > dy - dw && fi < dy + dw)
    if(x >= area.x && x <= area.z && fi >= area.y && fi <= area.w)
    {
        f = save_inside;
    }
    else
    {
        f = !save_inside;
    }

    wavelet[ x + fi*numElem.width ].x *= f;
    wavelet[ x + fi*numElem.width ].y *= f;
}
