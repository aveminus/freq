#include "hip/hip_runtime.h"
#include "neat_math.h"

__global__ void kernel_memset_fix(
        float2* p,
        unsigned N)
{
    const unsigned
            i = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;

    if (i>=N)
        return;

    p[i] = make_float2(0,0);
}


void cudaMemsetFix(void* p, unsigned N)
{
    if (N%sizeof(float2))
    {
        hipMemset(p, 0, N);
        return;
    }

    N /= sizeof(float2);

    dim3 block(64,1,1);
    dim3 grid( int_div_ceil(N, block.x), 1, 1);

    unsigned L = 32768;
    if(grid.x>L) {
        cudaMemsetFix( ((char*)p) + block.x*L, (N-block.x*L)*sizeof(float2));
        grid.x = L;
    }

    kernel_memset_fix<<<grid, block>>>((float2*)p, N);
}
